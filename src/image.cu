#include "hip/hip_runtime.h"
#include "image2.cuh"

using namespace CUrnel;

template<>
__device__ float4 Image<float4, R3>::sample( R3::ScalarType ) { return make_float4(0,0,0,0); };

template<>
__device__ float Image<float, R2>::sample( R2::ScalarType ) { return 0.0f; };