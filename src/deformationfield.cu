#include "hip/hip_runtime.h"

#include <iostream>
#include <iomanip>

#include "deformationfield.cuh"

using namespace std;
using namespace thrust;

using namespace CUrnel;

template<typename D>
size_t getFieldLength( typename D::OrdinalType _size );

template<>
size_t getFieldLength<R2>( R2::OrdinalType _size )
{
  return _size.x * _size.y;
}


template<typename T, typename D>
DeformationField<T,D>::DeformationField( typename D::OrdinalType _size ) :
  extent (_size)
{
  size_t size = getFieldLength<D>(_size);
  hData.resize(size);
  dData.resize(size);
  dData[6*3 + 2] = make_float2(10,10);
}

template<typename D>
dim3 getBlocksHelper(D extent);

template<>
dim3 getBlocksHelper<R2::OrdinalType>(R2::OrdinalType extent)
{
  return dim3(extent.y,1,1);
}

template<typename T, typename D>
dim3 DeformationField<T,D>::getBlocks()
{
  return getBlocksHelper(extent);
}


template<typename D>
dim3 getThreadsHelper(D extent);

template<>
dim3 getThreadsHelper<R2::OrdinalType>(R2::OrdinalType extent)
{
  return dim3(extent.x,1,1);
}

template<typename T, typename D>
dim3 DeformationField<T,D>::getThreads()
{
  return getThreadsHelper(extent);
}


template<typename D>
__device__ int getIndexHelper(D pos, D extent, bool limit = true);

template<>
__device__ int getIndexHelper<R2::OrdinalType>(R2::OrdinalType pos, R2::OrdinalType extent, bool limit)
{
  int& x = pos.x;
  int& y = pos.y;
  if( limit )
    if(y < 1 || y > extent.y - 2 || x < 1 || x > extent.x -2 ) return -1;
  int id = y * extent.y + x;
  return id;
}

template<typename D>
__device__ D getPositionFromBlocksAndThreads();

template<>
__device__ R2::OrdinalType getPositionFromBlocksAndThreads<R2::OrdinalType>()
{
  return make_int2(threadIdx.x, blockIdx.x);
}

template<typename T, typename D>
__device__ int DeformationField<T,D>::getIndexFromBlocksAndThreads()
{
  typename D::OrdinalType pos = getPositionFromBlocksAndThreads<D::OrdinalType>();
  return getIndexHelper( pos, extent, true );
}

template<typename T>
__device__ void neighborsHelper( N3x3<T>* neighbors, R2::OrdinalType pos, R2::OrdinalType extent, T* data )
{
  neighbors->m11 = data[getIndexHelper(make_int2(pos.x - 1, pos.y - 1), extent, false)];
  neighbors->m21 = data[getIndexHelper(make_int2(pos.x + 0, pos.y - 1), extent, false)];
  neighbors->m31 = data[getIndexHelper(make_int2(pos.x + 1, pos.y - 1), extent, false)];

  neighbors->m12 = data[getIndexHelper(make_int2(pos.x - 1, pos.y + 0), extent, false)];
  neighbors->m22 = data[getIndexHelper(make_int2(pos.x + 0, pos.y + 0), extent, false)];
  neighbors->m32 = data[getIndexHelper(make_int2(pos.x + 1, pos.y + 0), extent, false)];

  neighbors->m13 = data[getIndexHelper(make_int2(pos.x - 1, pos.y + 1), extent, false)];
  neighbors->m23 = data[getIndexHelper(make_int2(pos.x + 0, pos.y + 1), extent, false)];
  neighbors->m33 = data[getIndexHelper(make_int2(pos.x + 1, pos.y + 1), extent, false)];


}

template<typename T, typename D>
__device__ typename NeightborType<T,D>::type DeformationField<T,D>::getNeighbors( T* data )
{
  typename NeightborType<T,D>::type neighbors;
  typename D::OrdinalType pos = getPositionFromBlocksAndThreads<D::OrdinalType>();
  neighborsHelper(&neighbors, pos, extent, data );

  return neighbors;
}



template<typename T, typename D>
ostream& DeformationField<T,D>::print( ostream &out ) const
{
  hData = dData;	
  /*for( int j = 0; j < field.h; j++ )
  {
    for( int i = 0; i < field.w; i++ )
    {
      out << std::setw( 4 ) << field.hData[i + j*field.w] << " ";
    }
    out << "\n";
  }*/
  for(int i = 0; i < hData.size(); i++ )
  {
    out << setw(8) << hData[i].x << " ";
  }
  out << endl;
  return out;
}

template<>
ostream& DeformationField<float2,R2>::print( ostream &out ) const
{
  hData = dData;	
  /*for( int j = 0; j < field.h; j++ )
  {
    for( int i = 0; i < field.w; i++ )
    {
      out << std::setw( 4 ) << field.hData[i + j*field.w] << " ";
    }
    out << "\n";
  }*/
  out << endl;
  for(int i = 0; i < extent.y; i++ )
  {
    for(int j = 0; j < extent.x; j++ )
    {
      out << setw(8) << hData[i*extent.y + j].x << " ";
    }
    out << endl;
  }
  out << endl;
  return out;
}


template class DeformationField<float2, R2>;
