#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>

#include "image.cuh"
#include "image2.cuh"
#include "deformationfield.cuh"

#include "diffusionKernel.cuh"
#include "diffusion.cuh"

using namespace std;
using namespace CUrnel;

void showField( Field2D<float>& A )
{
	cout << A << endl;
	return;
}

int methodA(void)
{

	Field2D<float> A1 = Field2D<float>(6, 6);
	Field2D<float> A2 = Field2D<float>(6, 6);

	A1.rand();
    
    cout << "A1 (old):" << endl;
    showField( A1 );

	float alpha = -0.1;

	for( int i = 0; i < 200; i++ )
	{
		applyConvolution_diffusionKernel( A1, A2, alpha );
		A1.swapWith( A2 );
	}
   	cout << "A1 (new):" << endl;
	showField( A1 );



    return 0;
}

int methodB(void)
{
  int2 size = make_int2(6,6);
  Image<float, R2> r;
  Image<float, R2> t;

  DeformationField<float2, R2> phi1(size);
  DeformationField<float2, R2> phi2(size);

  cout << "phi1:" << phi1 << endl;
  cout << "phi2:" << phi2 << endl;

  apply_diffusion( phi1, phi2, -0.1 );

  cout << "phi1:" << phi1 << endl;
  cout << "phi2:" << phi2 << endl;

  apply_diffusion( phi2, phi1, -0.1 );

  cout << "phi2:" << phi2 << endl;
  cout << "phi1:" << phi1 << endl;

  return 0;
}

int main(void)
{
  methodA();
  methodB();
  return 0;
}


/*
int main()
{

  imiCudaImage<float4, R3> image0;
  imiCudaDeformableImage<imiLocalDeformationField, float4, R3> image1a;
  imiCudaDeformableImage<imiDeformationField<float4, R3>, float4, R3> image1b;
  imiCuda4DMotionDeformable<imiLocalDeformationField, float4> image2a;
  imiCuda4DMotionDeformable<imiPartialDeformationField, float4> image2b;

  std::cout<<"testing"<<std::endl;
  render( image0 );
  render( image1a );
  render( image1b );
  render( image2a );
  render( image2b );

  imiLocalDeformationField& dfield = image1a.getDeformationField();
  //applyDiffusion( dfield );

  //imiCudaImage<float, R2> r;
  //imiCudaImage<float, R2> t;
  imiDeformationField<float, R2> phi;
  imiDeformationField<float, R3> phi3D;

  applyDiffusion( phi );
  applyDiffusion( phi3D );

}*/
